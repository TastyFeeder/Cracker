#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>


#define TEST_LEN_LINIT 20
#define CHARSET_LEN_LIMIT 100

#define CONST_CHARSET "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789_!?"
#define CONST_CHARSET_LENGTH (sizeof(CONST_CHARSET) - 1)

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 8

#define TOTAL_BLOCKS 65535UL
//#define TOTAL_BLOCKS 32768UL
#define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 128UL

#include "md5.cu"


uint8_t word_length;

char word[TEST_LEN_LINIT];
char charset[CHARSET_LEN_LIMIT];
char cracked[TEST_LEN_LINIT];

__device__ char device_charset[CHARSET_LEN_LIMIT];
__device__ char devicecracked[TEST_LEN_LINIT];

__device__ int get_next_dev(uint8_t* length, char* word, uint64_t increment){
    uint64_t idx = 0;
    uint64_t add = 0;
  
    while(increment > 0 && idx < TEST_LEN_LINIT)
    {
        if(idx >= *length && increment > 0)
        {
              increment--;
        }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
    }
  
    if(idx > *length)
    {
        *length = idx;
    }
  
    if(idx > CONST_WORD_LENGTH_MAX)
    {
        return 0;
    }

    return 1;
}

int get_next(uint8_t* length, char* word, uint64_t increment){
    uint64_t idx = 0;
    uint64_t add = 0;
  
    while(increment > 0 && idx < TEST_LEN_LINIT)
    {
        if(idx >= *length && increment > 0)
        {
              increment--;
        }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
    }
  
    if(idx > *length)
    {
        *length = idx;
    }
  
    if(idx > CONST_WORD_LENGTH_MAX)
    {
        return 0;
    }

    return 1;
}


__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04){
  //uint64_t idx = ((gridDim.x * blockIdx.y + blockIdx.x )* blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  uint64_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  
  /* Shared variables */
  __shared__ char sharedCharset[CHARSET_LEN_LIMIT];
  
  /* Thread variables */
  char threadCharsetWord[TEST_LEN_LINIT];
  char threadTextWord[TEST_LEN_LINIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;
  
  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, TEST_LEN_LINIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, device_charset, sizeof(uint8_t) * CHARSET_LEN_LIMIT);
  
  /* Increment current word by thread index */
  get_next_dev(&threadWordLength, threadCharsetWord, idx);
  
  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }
    
    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);   

    if(threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04){
      memcpy(devicecracked, threadTextWord, threadWordLength);
    }
    
    if(!get_next_dev(&threadWordLength, threadCharsetWord, 1)){
      break;
    }
  }
}



int main(int argc ,char *argv[])
{
    int opt = 0;   
    uint32_t md5Hash[4];
    /* Amount of available devices */
    int devices;
    hipGetDeviceCount(&devices);
  
    /* Sync type */
    hipSetDeviceFlags(hipDeviceScheduleSpin);
  
    /* Display amount of devices */
    printf("Notice: %d device(s) found\n",devices);
    while ((opt = getopt(argc, argv, "mht:f:")) != -1) {
        switch (opt) {
        case 'h':
            printf("input---> %s\n",argv[2]);
    // Get input md5 hex
            for(uint8_t i = 0; i < 4; i++)
            {
                char tmp[16];
    
                strncpy(tmp, argv[2] + i * 8, 8);
                sscanf(tmp, "%x", &md5Hash[i]);   
                md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
            }
            break;
        default:
            printf("Use -h to input hex of MD5 hash\n");
            exit(EXIT_FAILURE);
        }
    }
    //
    if(argc == 1)
    {
        printf("Use -h to input hex of MD5 hash\n");
        exit(EXIT_FAILURE);
    }
    // set default value
    for(uint8_t i = 0; i < TEST_LEN_LINIT; i++)
    {
        word[i] = 0;
        cracked[i] = 0;
    }
    for(uint8_t i = 0; i < CONST_CHARSET_LENGTH; i++)
    {
        charset[i] = CONST_CHARSET[i];
    }
    word_length = CONST_WORD_LENGTH_MIN;
    
    hipSetDevice(0);
    
    char **words;
    words = (char**)malloc(sizeof(char*) * devices);
    
    hipSetDeviceFlags(hipDeviceScheduleYield);
    /* Time */
    hipEvent_t clockBegin;
    hipEvent_t clockLast;

    hipEventCreate(&clockBegin);
    hipEventCreate(&clockLast);
    hipEventRecord(clockBegin, 0);

    for(int device = 0; device < devices; device++)
    {
        hipSetDevice(device);
    
    /* Copy to each device */
        hipMemcpyToSymbol(HIP_SYMBOL(device_charset), charset, sizeof(uint8_t) * CHARSET_LEN_LIMIT, 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(devicecracked), cracked, sizeof(uint8_t) * TEST_LEN_LINIT, 0, hipMemcpyHostToDevice);
    
    /* Allocate on each device */
        hipMalloc((void**)&words[device], sizeof(uint8_t) * TEST_LEN_LINIT);
    }
    while(true)
    {
        int result = 0;
        int found = 0;
        for(int device = 0; device < devices; device++)
        {
            hipSetDevice(device);
            hipMemcpy(words[device], word, sizeof(uint8_t) * TEST_LEN_LINIT, hipMemcpyHostToDevice);
            // Kernel function
            md5Crack<<<TOTAL_BLOCKS , TOTAL_THREADS>>>(word_length, words[device], md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);
            result = get_next(&word_length, word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
            
        }
        char now[TEST_LEN_LINIT];
        for(int i = 0 ; i < word_length ; i++)
        {
            now[i] = charset[word[i]];
        }
        printf("Now testing :");
        for(int i = 0; i < word_length; i++)
        {
            printf("%c",now[i]);        
        }
        printf("(%d)\n",word_length);
        
        for(int device = 0; device < devices; device++){
            hipSetDevice(device);

            /* Synchronize now */
            hipDeviceSynchronize();

            /* Copy result */
            hipMemcpyFromSymbol(cracked, HIP_SYMBOL(devicecracked), sizeof(uint8_t) * TEST_LEN_LINIT, 0, hipMemcpyDeviceToHost); 

            /* Check result */
            if(found = *cracked != 0)
            {     
                    printf("Notice: cracked %s\n",cracked);
                    break;
            }
        }
        
        if(!result || found){
            if(!result && !found){
                    printf("Notice: found nothing (host)");
            }

            break;
        }

    }    
    for(int device = 0; device < devices; device++){
        hipSetDevice(device);

        /* Free on each device */
        hipFree((void**)words[device]);
    }
    /* Free array */
    free(words);

    /* Main device */
    hipSetDevice(0);

    float milliseconds = 0;

    hipEventRecord(clockLast, 0);
    hipEventSynchronize(clockLast);
    hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

    printf("Notice: computation time %f ms\n",milliseconds);

    hipEventDestroy(clockBegin);
    hipEventDestroy(clockLast);
    
    return 0;
}

